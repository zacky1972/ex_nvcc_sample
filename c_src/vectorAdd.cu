#include <stdint.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const int32_t *A, const int32_t *B, int32_t *C, uint64_t numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

#ifdef __cplusplus
extern "C" {
#endif
bool add_s32_cuda(const int32_t *h_A, const int32_t *h_B, int32_t *h_C, uint64_t numElements)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    
    // compute numElements
    int size = numElements * sizeof(int32_t);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        return false;
    }

    // Allocate the device input vector A
    int32_t *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        // fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
        //    cudaGetErrorString(err));
        return false;
    }

    // Allocate the device input vector B
    int32_t *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess) {
        // fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
        //    cudaGetErrorString(err));
        return false;
    }


    // Allocate the device input vector C
    int32_t *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess) {
        // fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
        //    cudaGetErrorString(err));
        return false;
    }

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in
    // device memory
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        //fprintf(stderr,
        //    "Failed to copy vector A from host to device (error code %s)!\n",
        //    cudaGetErrorString(err));
        return false;
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        //fprintf(stderr,
        //    "Failed to copy vector B from host to device (error code %s)!\n",
        //    cudaGetErrorString(err));
        return false;
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
    //     threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
        // fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
        //     cudaGetErrorString(err));
        return false;
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        // fprintf(stderr,
        //     "Failed to copy vector C from device to host (error code %s)!\n",
        //     cudaGetErrorString(err));
        return false;
    }

    // Free device global memory
    err = hipFree(d_A);
    if (err != hipSuccess) {
        // fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
        //     cudaGetErrorString(err));
        return false;
    }
    err = hipFree(d_B);
    if (err != hipSuccess) {
        // fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
        //     cudaGetErrorString(err));
        return false;
    }
    err = hipFree(d_C);
    if (err != hipSuccess) {
        // fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
        //     cudaGetErrorString(err));
        return false;
    }

    return true;
}
#ifdef __cplusplus
}
#endif